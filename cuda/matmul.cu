
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_THREADS_PER_BLOCK 256     // could be 1024 at most
#define SQRT_MAX_THREADS_PER_BLOCK 16 //

// stole this from StackOverflow to handle errors
#define gpuErrchk(ans)                                                         \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

typedef struct {
  float prep_time;
  float run_time;
  char *calling_fn;

} TestResult;

void printDim3(dim3 grid, dim3 block) {
  printf("Grid: (%d, %d, %d) blocks.\nBlocks: (%d, %d, %d) threads.\n", grid.x,
         grid.y, grid.z, block.x, block.y, block.z);
}

void printTestResult(TestResult tr) {
  printf("%s:\n\tPrep: %.2fus\n\tRun : %.2fus\n", tr.calling_fn, tr.prep_time,
         tr.run_time);
}

bool checkMatrix(int *arr, int n) {
  for (int i = 0; i < n * n; i++) {
    if (arr[i] != n) {
      printf("%d!-\n\n", arr[i]);
      return false;
    }
  }
  return true;
}

void printMatrix(int *arr, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      printf("%5d", arr[i * n + j]);
    }
    printf("\n");
  }
  printf("\n");
}

dim3 calcGridSize(int n, dim3 block) {
  int gridDimX = (n + block.x - 1) / block.x;
  int gridDimY = (n + block.y - 1) / block.y;
  return dim3(gridDimX, gridDimY, 1);
}

dim3 calcBlockSize(int n) {
  int blockDimX =
      (n < SQRT_MAX_THREADS_PER_BLOCK) ? n : SQRT_MAX_THREADS_PER_BLOCK;
  int blockDimY =
      (n < SQRT_MAX_THREADS_PER_BLOCK) ? n : SQRT_MAX_THREADS_PER_BLOCK;
  return dim3(blockDimX, blockDimY, 1);
}

__global__ void transposeKernel(int n, int *input, int *output) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    output[col * n + row] = input[row * n + col];
  }
}

__global__ void coalescedOneDimKernel(int n, int *a, int *b, int *c) {
  const int x = blockIdx.x * blockDim.x + (threadIdx.x / blockDim.x);
  const int y = blockIdx.y * blockDim.y + (threadIdx.x % blockDim.y);

  if (x == 0 && y == 0) {
    printf("blockDim.x = %d, blockDim.y = %d\n", blockDim.x, blockDim.y);
    printf("(%d,%d), ", x, y);
  }
  // printf("(%d,%d), ", x, y);

  if (x < n && y < n) {
    int sum = 0;
    for (int k = 0; k < n; k++) {
      sum += a[x * n + k] * b[k * n + y];
    }
    c[x * n + y] = sum;
    // if (sum != n) printf("Error calculating from coalesced\tn = %d sum =
    // %d\n", n, sum);
  }
}

TestResult coalescedOneDim(int n) {
  hipEvent_t prep, start, end;
  hipEventCreate(&prep);
  hipEventCreate(&start);
  hipEventCreate(&end);
  float ms_prep, ms_run;

  hipEventRecord(prep);

  size_t capacity = n * n * sizeof(int);

  int *a, *b, *c;
  a = (int *)malloc(capacity);
  b = (int *)malloc(capacity);
  c = (int *)malloc(capacity);

  for (int i = 0; i < n * n; i++) {
    a[i] = 1;
    b[i] = 1;
  }

  int *dev_a, *dev_b, *dev_c;
  hipMalloc(&dev_a, capacity);
  hipMalloc(&dev_b, capacity);
  hipMalloc(&dev_c, capacity);

  gpuErrchk(hipMemcpy(dev_a, a, capacity, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_b, b, capacity, hipMemcpyHostToDevice));

  dim3 dimBlock = calcBlockSize(n);
  dim3 dimGrid = calcGridSize(n, dimBlock);
  printDim3(dimGrid, dimBlock);

  hipEventRecord(start);

  coalescedOneDimKernel<<<dimGrid, dimBlock>>>(n, dev_a, dev_b, dev_c);
  gpuErrchk(hipPeekAtLastError());

  hipMemcpy(c, dev_c, capacity, hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&ms_run, start, end);
  hipEventElapsedTime(&ms_prep, prep, start);

  TestResult tr;
  tr.calling_fn = "coalescedOneDim";
  tr.prep_time = ms_prep * 1000;
  tr.run_time = ms_run * 1000;

  free(a);
  free(b);
  free(c);

  return tr;
}

__global__ void transOneDimKernel(int n, int *a, int *b_t, int *c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    int sum = 0;
    for (int k = 0; k < n; k++) {
      sum += a[row * n + k] * b_t[col * n + k];
    }
    c[row * n + col] = sum;
  }
}

TestResult transOneDim(int n) {
  hipEvent_t prep, start, end;
  hipEventCreate(&prep);
  hipEventCreate(&start);
  hipEventCreate(&end);
  float ms_prep, ms_run;

  hipEventRecord(prep);

  size_t capacity = n * n * sizeof(int);

  int *a, *b, *c;
  a = (int *)malloc(capacity);
  b = (int *)malloc(capacity);
  c = (int *)malloc(capacity);

  for (int i = 0; i < n * n; i++) {
    a[i] = 1;
    b[i] = 1;
  }

  int *dev_a, *dev_b, *dev_b_t, *dev_c;
  hipMalloc(&dev_a, capacity);
  hipMalloc(&dev_b, capacity);
  hipMalloc(&dev_b_t, capacity);
  hipMalloc(&dev_c, capacity);

  gpuErrchk(hipMemcpy(dev_a, a, capacity, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_b, b, capacity, hipMemcpyHostToDevice));
  // gpuErrchk(cudaMemcpy(dev_c, c, capacity, cudaMemcpyHostToDevice));

  dim3 dimBlock = calcBlockSize(n);
  dim3 dimGrid = calcGridSize(n, dimBlock);

  hipEventRecord(start);

  transposeKernel<<<dimGrid, dimBlock>>>(n, dev_b, dev_b_t);
  gpuErrchk(hipPeekAtLastError());

  transOneDimKernel<<<dimGrid, dimBlock>>>(n, dev_a, dev_b_t, dev_c);
  gpuErrchk(hipPeekAtLastError());

  hipMemcpy(c, dev_c, capacity, hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_b_t);
  hipFree(dev_c);

  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&ms_run, start, end);
  hipEventElapsedTime(&ms_prep, prep, start);

  TestResult tr;
  tr.calling_fn = "transOneDim";
  tr.prep_time = ms_prep * 1000;
  tr.run_time = ms_run * 1000;

  free(a);
  free(b);
  free(c);

  return tr;
}

__global__ void naiveOneDimKernel(int n, int *a, int *b, int *c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    int sum = 0;
    for (int k = 0; k < n; k++) {
      sum += a[row * n + k] * b[k * n + col];
    }
    c[row * n + col] = sum;
  }
}

TestResult naiveOneDim(int n) {
  hipEvent_t prep, start, end;
  hipEventCreate(&prep);
  hipEventCreate(&start);
  hipEventCreate(&end);
  float ms_prep, ms_run;

  hipEventRecord(prep);

  size_t capacity = n * n * sizeof(int);

  int *a, *b, *c;
  a = (int *)malloc(capacity);
  b = (int *)malloc(capacity);
  c = (int *)malloc(capacity);

  for (int i = 0; i < n * n; i++) {
    a[i] = 1;
    b[i] = 1;
  }

  int *dev_a, *dev_b, *dev_c;
  hipMalloc(&dev_a, capacity);
  hipMalloc(&dev_b, capacity);
  hipMalloc(&dev_c, capacity);

  gpuErrchk(hipMemcpy(dev_a, a, capacity, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_b, b, capacity, hipMemcpyHostToDevice));
  // gpuErrchk(cudaMemcpy(dev_c, c, capacity, cudaMemcpyHostToDevice));

  dim3 dimBlock(calcBlockSize(n));
  dim3 dimGrid(calcGridSize(n, dimBlock));

  hipEventRecord(start);
  naiveOneDimKernel<<<dimGrid, dimBlock>>>(n, dev_a, dev_b,
                                           dev_c); // block, threads per block
  gpuErrchk(hipPeekAtLastError());

  // cudaMemcpy(a, dev_a, capacity, cudaMemcpyDeviceToHost);
  // cudaMemcpy(b, dev_b, capacity, cudaMemcpyDeviceToHost);
  hipMemcpy(c, dev_c, capacity, hipMemcpyDeviceToHost);

  if (!checkMatrix(c, n)) {
    fprintf(stderr, "ERROR: MATRIX MULTIPLICATION DIDN'T WORK!!!!!\n");
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventRecord(end);
  hipEventSynchronize(end);

  hipEventElapsedTime(&ms_run, start, end);
  hipEventElapsedTime(&ms_prep, prep, start);

  free(a);
  free(b);
  free(c);

  TestResult tr;
  tr.calling_fn = "naiveOneDim";
  tr.prep_time = ms_prep * 1000;
  tr.run_time = ms_run * 1000;

  return tr;
}

int main(int argc, char **argv) {
  if (argc < 2) {
    fprintf(stderr, "Usage: ./a.out N, where N is the matrix size expressed as "
                    "2 ** N. Exiting\n");
    return EXIT_FAILURE;
  }

  int n_exp = atoi(argv[1]);
  int n = 1 << n_exp;

  TestResult naive_one_dim = naiveOneDim(n);
  printTestResult(naive_one_dim);

  TestResult transOneDim_tr = transOneDim(n);
  printTestResult(transOneDim_tr);

  TestResult coalesced_one_dim = coalescedOneDim(n);
  printTestResult(coalesced_one_dim);

  return EXIT_SUCCESS;
}
